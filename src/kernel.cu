#include "hip/hip_runtime.h"
#include<stdio.h>
#include"hip/hip_runtime.h"

__global__ void test_print_kernel(const float* pdata, int ndata){
    int idx = threadIdx.x + blockIdx.x;
    int index = ((((blockIdx.z*gridDim.y+blockIdx.y)*gridDim.x+blockIdx.x)*blockDim.z+threadIdx.z)*blockDim.y+threadIdx.y)*blockDim.x+threadIdx.x
    
    /*          dim             idx
            gridDim.z       blockIdx.z
            gridDim.y       blockIdx.y
            gridDim.x       blockIdx.x
            blockDim.z      threadIdx.z
            blockDim.y      threadIdx.y
            blockDim.x      threadIdx.x

            idx = (gridDim.x*blockIdx.y)+gridDim.y
    */
    printf("Element[%d] = %f, threadIdx.x=%d, blockIdx.x=%d, blockDim.x=%d\n", idx, pdata[idx], threadIdx.x, blockIdx.x, blockDim.x);


}

void test_print(const float* pdata, int ndata){

    // <<<gridDim, blockDim, bytes_of_shared_memory, stream>>>
    test_print_kernel<<<1,ndata,0,nullptr>>>(pdata,ndata);

    hipError_t code = hipPeekAtLastError();
    if(code!=hipSuccess){
        const char* err_name = hipGetErrorName(code);
        const char* err_message = hipGetErrorString(code);
        printf("kernel error %s:%d  test_print_kernel failed. \n  code = %s, message = %s\n", __FILE__, __LINE__, err_name, err_message);   

    }

}
