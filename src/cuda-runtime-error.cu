#include "hip/hip_runtime.h"
#include<stdio.h>

#include"hip/hip_runtime.h"
using namespace std;

__global__ void func(float* ptr){
    int tid = blockIdx.x*blockDim.x+threadIdx.x;
    if(tid == 999)
        ptr[999] = 5;
}


int main(){
    float* ptr = nullptr;
    func<<<100,10>>>(ptr);
    hipError_t code = hipPeekAtLastError();
    cout<<hipGetErrorString(code)<<endl;

    hipError_t code1 = hipDeviceSynchronize();
    cout<<hipGetErrorString(code1)<<endl;

    float* new_ptr = nullptr;
    auto code2 = hipMalloc(&new_ptr,100*sizeof(float));
    cout<<hipGetErrorString(code2)<<endl;
    return 0;

}